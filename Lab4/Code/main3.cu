
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



 __global__ void histo_kernel_2 (unsigned char *buffer, int img_w, int img_h, int *histo)
{
    int id_x = blockIdx.x * blockDim.x + threadIdx.x ;
    int id_y = blockIdx.y * blockDim.y + threadIdx.y ;

    atomicAdd (&histo[buffer[id_y*img_w + id_x]] , 1 );
}





__global__ void histo_kernel ( unsigned char *buffer, long size, int *histo )
{
    __shared__ int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
    while (i < size)
    {
        atomicAdd( &temp[buffer[i]], 1);
        i += offset;
    }
    __syncthreads();


    atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}




typedef struct{
    int w;
    int h;
    unsigned char * img;
} PGM_IMG;    


PGM_IMG read_pgm(const char * path);
void write_pgm(PGM_IMG img, const char * path);
void free_pgm(PGM_IMG img);
void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin);
void histogram_equalization(unsigned char * img_out, unsigned char * img_in, int * hist_in, int img_size, int nbr_bin);
void run_cpu_gray_test(PGM_IMG img_in, char *out_filename);

int main(int argc, char *argv[]){
    PGM_IMG img_ibuf_g;

	if (argc != 3) {
		printf("Run with input file name and output file name as arguments\n");
		exit(1);
	}
	
    printf("Running contrast enhancement for gray-scale images.\n");
    img_ibuf_g = read_pgm(argv[1]);
    run_cpu_gray_test(img_ibuf_g, argv[2]);
    free_pgm(img_ibuf_g);

	return 0;
}



void run_cpu_gray_test(PGM_IMG img_in, char *out_filename)
{
    //unsigned int timer = 0;
    PGM_IMG img_obuf;

    struct timespec  tv1, tv2, tv3, tv4;
    

    printf("Starting CPU processing...\n");
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);

    PGM_IMG result;
    int hist[256];
    
    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

    clock_gettime(CLOCK_MONOTONIC_RAW, &tv3);
    
    histogram(hist, img_in.img, img_in.h * img_in.w, 256);

    clock_gettime(CLOCK_MONOTONIC_RAW, &tv4);

    histogram_equalization(result.img,img_in.img,hist,result.w*result.h, 256);

    //img_obuf = result;
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);

    printf ("Time for the CPU: \t\t\t%g s\n",
            (double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
           (double) (tv2.tv_sec - tv1.tv_sec));

    printf ("Time to generate Hist in CPU: \t\t%g s\n",
            (double) (tv4.tv_nsec - tv3.tv_nsec) / 1000000000.0 +
           (double) (tv4.tv_sec - tv3.tv_sec));

    printf ("Time for Hist Equalization in CPU: \t%g s\n",
            (double) (tv2.tv_nsec - tv4.tv_nsec) / 1000000000.0 +
           (double) (tv2.tv_sec - tv4.tv_sec));


    printf("Starting GPU processing...\n");

    PGM_IMG result2;
    int histo[256];
    
    result2.w = img_in.w;
    result2.h = img_in.h;
    result2.img = (unsigned char *)malloc(result2.w * result2.h * sizeof(unsigned char));

    hipEvent_t start, stop;
    gpuErrchk( hipEventCreate( &start ) );
    gpuErrchk( hipEventCreate( &stop ) );
    gpuErrchk( hipEventRecord( start, 0 ) );


    unsigned char *dev_buffer;
    int *dev_histo;
    gpuErrchk( hipMalloc( (void**)&dev_buffer, img_in.w * img_in.h * sizeof(unsigned char) ) );
    gpuErrchk( hipMemcpy( dev_buffer, img_in.img, img_in.w * img_in.h * sizeof(unsigned char), hipMemcpyHostToDevice ) );
    gpuErrchk( hipMalloc( (void**)&dev_histo,256 * sizeof( int ) ) );
    gpuErrchk( hipMemset( dev_histo, 0,256 * sizeof( int ) ) );

    hipDeviceProp_t prop;
    gpuErrchk( hipGetDeviceProperties( &prop, 0 ) );
    int blocks = prop.multiProcessorCount;
    printf("Blocks = %d\n", blocks);

    dim3 grid ((img_in.w)/8, (img_in.h)/8);
    dim3 block (8, 8);

    histo_kernel <<<blocks*256, 256>>>( dev_buffer, img_in.w * img_in.h, dev_histo );
    //histo_kernel <<<(img_in.w * img_in.h)/(256*8), 256>>>( dev_buffer, img_in.w * img_in.h, dev_histo );
    //histo_kernel_2 <<<grid, block>>>( dev_buffer, img_in.w, img_in.h , dev_histo );

    gpuErrchk ( hipMemcpy( histo, dev_histo, 256 * sizeof( int ), hipMemcpyDeviceToHost ) );


    gpuErrchk ( hipEventRecord( stop, 0 ) );
    gpuErrchk ( hipEventSynchronize( stop ) );
    float elapsedTime;
    gpuErrchk ( hipEventElapsedTime( &elapsedTime,start, stop ) );
    printf( "Time to generate Histo in GPU: \t\t%f s\n", elapsedTime / 1000 );


    
    // verify that we have the same counts via CPU
    long int histoCount = 0;
    for (int i=0; i<256; i++) {
        histoCount += hist[i];
    }
    printf( "Histogram Sum in CPU: %ld\n", histoCount );

    histoCount = 0;
    for (int i=0; i<256; i++) {
        histoCount += histo[i];
    }
    printf( "Histogram Sum in GPU: %ld\n", histoCount );

    

    gpuErrchk ( hipEventDestroy( start ) );
    gpuErrchk ( hipEventDestroy( stop ) );
    gpuErrchk ( hipFree( dev_histo ) ) ;
    gpuErrchk ( hipFree( dev_buffer ) ) ;



    histogram_equalization(result2.img,img_in.img,histo,result2.w*result2.h, 256);

    img_obuf = result2;



    write_pgm(img_obuf, out_filename);
    free_pgm(img_obuf);
}


PGM_IMG read_pgm(const char * path){
    FILE * in_file;
    char sbuf[256];
    
    
    PGM_IMG result;
    int v_max;//, i;
    in_file = fopen(path, "r");
    if (in_file == NULL){
        printf("Input file not found!\n");
        exit(1);
    }
    
    fscanf(in_file, "%s", sbuf); /*Skip the magic number*/
    fscanf(in_file, "%d",&result.w);
    fscanf(in_file, "%d",&result.h);
    fscanf(in_file, "%d\n",&v_max);
    printf("Image size: %d x %d\n", result.w, result.h);
    

    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

        
    fread(result.img,sizeof(unsigned char), result.w*result.h, in_file);    
    fclose(in_file);
    
    return result;
}

void write_pgm(PGM_IMG img, const char * path){
    FILE * out_file;
    out_file = fopen(path, "wb");
    fprintf(out_file, "P5\n");
    fprintf(out_file, "%d %d\n255\n",img.w, img.h);
    fwrite(img.img,sizeof(unsigned char), img.w*img.h, out_file);
    fclose(out_file);
}

void free_pgm(PGM_IMG img)
{
    free(img.img);
}

void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
    int i;
    for ( i = 0; i < nbr_bin; i++){
        hist_out[i] = 0;
    }

    for ( i = 0; i < img_size; i++){
        hist_out[img_in[i]] ++;
    }
}

void histogram_equalization(unsigned char * img_out, unsigned char * img_in, int * hist_in, int img_size, int nbr_bin){
    int *lut = (int *)malloc(sizeof(int)*nbr_bin);
    int i, cdf, min, d;
    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    while(min == 0){
        min = hist_in[i++];
    }
    d = img_size - min;
    for(i = 0; i < nbr_bin; i++){
        cdf += hist_in[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
        if(lut[i] < 0){
            lut[i] = 0;
        }    
    }
    
    /* Get the result image */
    for(i = 0; i < img_size; i++){
        if(lut[img_in[i]] > 255){
            img_out[i] = 255;
        }
        else{
            img_out[i] = (unsigned char)lut[img_in[i]];
        }       
    }
}