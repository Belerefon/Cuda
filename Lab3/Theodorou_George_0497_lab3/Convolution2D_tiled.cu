/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//unsigned int filter_radius;

#define filter_radius 16
#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005
#define TILE_WIDTH 16
#define TILE_W1 16
#define TILE_H1 16
#define TILE_W2 16
#define TILE_H2 128


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, int imageW, int imageH, int filterR);
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter, int imageW, int imageH, int filterR);

__constant__ float d_Filter[FILTER_LENGTH];

///Kernel gia efarmogh filtrou kata grammes
 __global__
 void convolutionRowGPU(float *d_Buffer, float *d_Input, int imageW, int imageH) {

	__shared__ float data[TILE_H1 * (TILE_W1 + 2*filter_radius)];

	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	int reg = (y + filter_radius) * (imageW + 2*filter_radius) + (x + filter_radius);
	int data_reg = threadIdx.x + threadIdx.y * (TILE_W1 + 2*filter_radius);

	data[data_reg] = d_Input[reg - filter_radius];
	data[data_reg + blockDim.x] = d_Input[reg];
	data[data_reg + 2*blockDim.x] = d_Input[reg + filter_radius];

	__syncthreads();

	float sum = 0;
	#pragma unroll
	for (int k = -filter_radius; k <= filter_radius; k++)
		sum += data[data_reg + filter_radius + k] * d_Filter[filter_radius - k];

	d_Buffer[reg] = sum;
 }

//Kernel gia efarmogh filtrou kata sthles
 __global__
 void convolutionColumnGPU(float *d_Output, float *d_Buffer, int imageW, int imageH) {

	__shared__ float data[TILE_W2 * (TILE_H2 + 2*filter_radius)];

	int y = blockIdx.y * 8*blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	int reg = (y + filter_radius) * (imageW + 2*filter_radius) + (x + filter_radius);
	int data_reg = threadIdx.x + threadIdx.y * TILE_W2;
	
	data[data_reg] = d_Buffer[reg - (imageW + 2*filter_radius) * filter_radius];
	data[data_reg + blockDim.y * TILE_W2] = d_Buffer[reg];
	data[data_reg + 2*blockDim.y * TILE_W2] = d_Buffer[reg + (imageW + 2*filter_radius) * filter_radius];
	data[data_reg + 3*blockDim.y * TILE_W2] = d_Buffer[reg + (imageW + 2*filter_radius) * (filter_radius + blockDim.x)];
	data[data_reg + 4*blockDim.y * TILE_W2] = d_Buffer[reg + (imageW + 2*filter_radius) * (filter_radius + 2*blockDim.x)];
	data[data_reg + 5*blockDim.y * TILE_W2] = d_Buffer[reg + (imageW + 2*filter_radius) * (filter_radius + 3*blockDim.x)];
	data[data_reg + 6*blockDim.y * TILE_W2] = d_Buffer[reg + (imageW + 2*filter_radius) * (filter_radius + 4*blockDim.x)];
	data[data_reg + 7*blockDim.y * TILE_W2] = d_Buffer[reg + (imageW + 2*filter_radius) * (filter_radius + 5*blockDim.x)];
	data[data_reg + 8*blockDim.y * TILE_W2] = d_Buffer[reg + (imageW + 2*filter_radius) * (filter_radius + 6*blockDim.x)];
	data[data_reg + 9*blockDim.y * TILE_W2] = d_Buffer[reg + (imageW + 2*filter_radius) * (filter_radius + 7*blockDim.x)];
	

	__syncthreads();

	float sum1, sum2, sum3, sum4, sum5, sum6, sum7, sum8;
	sum1 = sum2 = sum3 = sum4 = sum5 = sum5 = sum7 = sum8 = 0;
	#pragma unroll
	for (int k = -filter_radius; k <= filter_radius; k++) {
		sum1 += data[data_reg + (k + filter_radius) * TILE_W2] * d_Filter[filter_radius - k];
		sum2 += data[data_reg + (blockDim.y + k + filter_radius) * TILE_W2] * d_Filter[filter_radius - k];
		sum3 += data[data_reg + (2*blockDim.y + k + filter_radius) * TILE_W2] * d_Filter[filter_radius - k];
		sum4 += data[data_reg + (3*blockDim.y + k + filter_radius) * TILE_W2] * d_Filter[filter_radius - k];
		sum5 += data[data_reg + (4*blockDim.y + k + filter_radius) * TILE_W2] * d_Filter[filter_radius - k];
		sum6 += data[data_reg + (5*blockDim.y + k + filter_radius) * TILE_W2] * d_Filter[filter_radius - k];
		sum7 += data[data_reg + (6*blockDim.y + k + filter_radius) * TILE_W2] * d_Filter[filter_radius - k];
		sum8 += data[data_reg + (7*blockDim.y + k + filter_radius) * TILE_W2] * d_Filter[filter_radius - k];
		
	}

	d_Output[y * imageW + x] = sum1;
	d_Output[(y + blockDim.y) * imageW + x] = sum2;
	d_Output[(y + 2*blockDim.y) * imageW + x] = sum3;
	d_Output[(y + 3*blockDim.y) * imageW + x] = sum4;
	d_Output[(y + 4*blockDim.y) * imageW + x] = sum5;
	d_Output[(y + 5*blockDim.y) * imageW + x] = sum6;
	d_Output[(y + 6*blockDim.y) * imageW + x] = sum7;
	d_Output[(y + 7*blockDim.y) * imageW + x] = sum8;
	
}

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter,
                       int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

	//cudaDeviceSetCacheConfig(cudaFuncCachePreferShared);
	//cudaFuncSetCacheConfig(convolutionRowGPU, cudaFuncCachePreferShared);
	//cudaFuncSetCacheConfig(convolutionColumnGPU, cudaFuncCachePreferShared);

	//struct timespec  tv1, tv2;

    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
	*h_OutputGPU,
	*h_Input_padding,
	*d_Input,
	*d_Buffer,
	*d_Output;


    int imageW;
    int imageH;
    unsigned int i;

	//printf("Enter filter radius : ");
	//scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
	if ( imageW < FILTER_LENGTH ) {
		printf("Image size lower than %d\nTermatismos programmatos...\n", FILTER_LENGTH);
		return(2);
	}
	else if ( imageW % 2 != 0 ) {
		printf("Image size is not a power of two\nTermatismos programmatos...\n");
		return(3);
	}
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    //printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
	h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));

	h_Input_padding 	= (float *)malloc((imageW + 2*filter_radius) * (imageH + 2*filter_radius) * sizeof(float));

	//Elegxos apotelesmatwn twn malloc
	if (h_Filter == NULL || h_Input == NULL || h_Buffer == NULL || h_OutputCPU == NULL || h_OutputGPU == NULL || h_Input_padding == NULL ) {
		printf("Apotuxia Desmeushs mnhmhs \n Termatismos programmatos...\n");
		return(1);
	}

	//Desmeush mnhmhs gia to device
	printf("Allocating and initializing device arrays...\n");
	gpuErrchk( hipMalloc((void**)&d_Input, (imageW + 2*filter_radius) * (imageH + 2*filter_radius) * sizeof(float)) );
	gpuErrchk( hipMalloc((void**)&d_Buffer, (imageW + 2*filter_radius) * (imageH + 2*filter_radius) * sizeof(float)) );
	gpuErrchk( hipMalloc((void**)&d_Output, imageW * imageH * sizeof(float)) );



    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }


	//Topothethsh tou padding perimetrika ths eikonas eisodou
	for (i=0; i < (imageW + 2*filter_radius) * filter_radius; i++)
		h_Input_padding[i] = 0;

	int p = 0, k = 0;
	for (i = (imageW + 2*filter_radius) * filter_radius; i < (imageW + 2*filter_radius) * (filter_radius + imageW); i++ ) {
		if ( p < filter_radius || p >= filter_radius + imageW ) {
			h_Input_padding[i] = 0;
		}
		else {
			h_Input_padding[i] = h_Input[k];
			k++;
		}
		p++;
		if ( p == 2*filter_radius + imageW )
			p = 0;
	}

	for (i=(imageW + 2*filter_radius) * (filter_radius + imageW); i < (imageW + 2*filter_radius) * (2*filter_radius + imageW); i++)
		h_Input_padding[i] = 0;





	// To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
	/*printf("CPU computation...\n");

	clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);

	convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
	convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

	clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);
	*/


	//To parakatw einai to kommati pou xreiazetai gia thn ektelesh sthn GPU
	printf("GPU computation...\n");

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	dim3 dimGrid1(imageW/TILE_W1, imageH/TILE_H1);
	dim3 dimGrid2(imageW/TILE_W2, imageH/TILE_H2);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);


	//Metafora dedomenwn apo ton host pros to device
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_Filter), h_Filter, FILTER_LENGTH * sizeof(float)) );
	gpuErrchk( hipMemcpy(d_Input, h_Input_padding, (imageW + 2*filter_radius) * (2*filter_radius + imageW) * sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemset(d_Buffer, 0, (imageW + 2*filter_radius) * (2*filter_radius + imageW) * sizeof(float)) );


	hipEventRecord(start);
	//Kaloume ton prwto Kernel
	convolutionRowGPU<<<dimGrid1, dimBlock>>>(d_Buffer, d_Input, imageW, imageH);

	gpuErrchk( hipPeekAtLastError() );


	//Kaloume ton deutero Kernel
	convolutionColumnGPU<<<dimGrid2, dimBlock>>>(d_Output, d_Buffer, imageW, imageH);

	gpuErrchk( hipPeekAtLastError() );

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float GPUtime;
	hipEventElapsedTime(&GPUtime, start, stop);


	//Metafora apotelesmatos apo to device ston host
	gpuErrchk( hipMemcpy(h_OutputGPU, d_Output,  imageW * imageH * sizeof(float), hipMemcpyDeviceToHost) );

	
	// Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas

	/*for (i = 0; i < imageW * imageH; i++) {
		if (ABS(h_OutputCPU[i] - h_OutputGPU[i]) > accuracy) {
			printf("Sfalma akriveias \n Termatismos programmatos...\n");
			return(2);
		}
		//printf("h_OutputCPU[%d]: %f\nh_OutputGPU[%d]: %f\n\n", i, h_OutputCPU[i], i, h_OutputGPU[i]);
	}
	*/

	p = 0;
	int sum = 0;
	float max = accuracy;
	for (i = 0; i < imageW * imageH; i++) {
		if (ABS(h_OutputCPU[i] - h_OutputGPU[i]) > max ) {
			max = ABS(h_OutputCPU[i] - h_OutputGPU[i]);
			p = i;
		}
	}
	sum += max;
	printf("max: %f\n", max);
	printf("h_OutputCPU[%d]: %f\n",p, h_OutputCPU[p]);
	printf("h_OutputGPU[%d]: %f\n\n",p, h_OutputGPU[p]);


	//Ektypwsh xronwn
	//printf ("Time for the CPU: %10g s\n",
	//		(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
	//		(double) (tv2.tv_sec - tv1.tv_sec));
	printf("Time for the GPU: %f s\n", GPUtime / 1000 );


    // free all the allocated memory
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipFree(d_Output);

	free(h_OutputGPU);
	free(h_OutputCPU);
	free(h_Buffer);
	free(h_Input);
	free(h_Filter);
	free(h_Input_padding);

	// Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
	hipDeviceReset();


    return 0;
}
